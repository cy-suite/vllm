#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Adapted from https://github.com/IST-DASLab/marlin
 */

#include "gptq_marlin.cuh"

template <typename T> inline std::string str(T x) { return std::to_string(x); }

namespace gptq_marlin {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

__global__ void permute_cols_kernel(int4 const *__restrict__ a_int4_ptr,
                                    int const *__restrict__ perm_int_ptr,
                                    int4 *__restrict__ out_int4_ptr, int size_m,
                                    int size_k, int block_rows) {}

template <const int num_bits,        // number of bits used for weights
          const int threads,         // number of threads in a threadblock
          const int thread_m_blocks, // number of 16x16 blocks in the m
                                     // dimension (batchsize) of the threadblock
          const int thread_n_blocks, // same for n dimension (output)
          const int thread_k_blocks, // same for k dimension (reduction)
          const int stages, // number of stages for the async global->shared
                            // fetch pipeline
          const bool has_act_order,   // whether act_order is enabled
          const int group_blocks = -1 // number of consecutive 16x16 blocks with
                                      // a separate quantization scale
          >
__global__ void
Marlin(const int4 *__restrict__ A, // fp16 input matrix of shape mxk
       const int4 *__restrict__ B, // 4bit quantized weight matrix of shape kxn
       int4 *__restrict__ C,       // fp16 output buffer of shape mxn
       const int4 *__restrict__ scales_ptr, // fp16 quantization scales of shape
                                            // (k/groupsize)xn
       const int *__restrict__ g_idx,       // int32 group indices of shape k
       int num_groups, // number of scale groups per output channel
       int prob_m,     // batch dimension m
       int prob_n,     // output dimension n
       int prob_k,     // reduction dimension k
       int *locks      // extra global storage for barrier synchronization
) {}

} // namespace gptq_marlin

torch::Tensor gptq_marlin_gemm(torch::Tensor &a, torch::Tensor &b_q_weight,
                               torch::Tensor &b_scales, torch::Tensor &g_idx,
                               torch::Tensor &perm, torch::Tensor &workspace,
                               int64_t num_bits, int64_t size_m, int64_t size_n,
                               int64_t size_k, bool is_k_full) {
  TORCH_CHECK_NOT_IMPLEMENTED(false,
                              "marlin_gemm(..) requires CUDA_ARCH >= 8.0");
  return torch::empty({1, 1});
}

#else


  namespace fp16 {
#include "gptq_marlin_part.cu"
  } // namespace fp16


  namespace bf16 {
#define INCLUDE_GPT_MARGIN_BFLOAT16 1
#include "gptq_marlin_part.cu"
#undef INCLUDE_GPT_MARGIN_BFLOAT16
  } // namespace bf16

} // namespace gptq_marlin


torch::Tensor gptq_marlin_gemm(torch::Tensor &a, torch::Tensor &b_q_weight,
                               torch::Tensor &b_scales, torch::Tensor &g_idx,
                               torch::Tensor &perm, torch::Tensor &workspace,
                               int64_t num_bits, int64_t size_m, int64_t size_n,
                               int64_t size_k, bool is_k_full) {
  // Verify num_bits
  TORCH_CHECK(num_bits == 4 || num_bits == 8,
              "num_bits must be 4 or 8. Got = ", num_bits);
  int pack_factor = 32 / num_bits;

  // Verify A
  TORCH_CHECK(a.size(0) == size_m, "Shape mismatch: a.size(0) = ", a.size(0),
              ", size_m = ", size_m);
  TORCH_CHECK(a.size(1) == size_k, "Shape mismatch: a.size(1) = ", a.size(1),
              ", size_k = ", size_k);

  // Verify B
  TORCH_CHECK(size_k % gptq_marlin::tile_size == 0, "size_k = ", size_k,
              " is not divisible by tile_size = ", gptq_marlin::tile_size);
  TORCH_CHECK((size_k / gptq_marlin::tile_size) == b_q_weight.size(0),
              "Shape mismatch: b_q_weight.size(0) = ", b_q_weight.size(0),
              ", size_k = ", size_k, ", tile_size = ", gptq_marlin::tile_size);
  TORCH_CHECK(b_q_weight.size(1) % gptq_marlin::tile_size == 0,
              "b_q_weight.size(1) = ", b_q_weight.size(1),
              " is not divisible by tile_size = ", gptq_marlin::tile_size);
  int actual_size_n =
      (b_q_weight.size(1) / gptq_marlin::tile_size) * pack_factor;
  TORCH_CHECK(size_n == actual_size_n, "size_n = ", size_n,
              ", actual_size_n = ", actual_size_n);

  // Verify device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");

  TORCH_CHECK(b_scales.device().is_cuda(), "b_scales is not on GPU");
  TORCH_CHECK(b_scales.is_contiguous(), "b_scales is not contiguous");

  TORCH_CHECK(g_idx.device().is_cuda(), "g_idx is not on GPU");
  TORCH_CHECK(g_idx.is_contiguous(), "g_idx is not contiguous");

  TORCH_CHECK(perm.device().is_cuda(), "perm is not on GPU");
  TORCH_CHECK(perm.is_contiguous(), "perm is not contiguous");

  // Alloc buffers
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  torch::Tensor c = torch::empty({size_m, size_n}, options);
  torch::Tensor a_tmp = torch::empty({size_m, size_k}, options);

  // thread_k: `k` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_k = -1;
  // thread_n: `n` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_n = -1;
  // sms: number of SMs to use for the kernel (can usually be left as auto -1)
  int sms = -1;

  // Verify g_idx and perm
  TORCH_CHECK((g_idx.size(0) == 0 && perm.size(0) == 0) ||
                  (g_idx.size(0) == size_k && perm.size(0) == size_k),
              "Unexpected g_idx.size(0) = ", g_idx.size(0),
              " and perm.size(0) = ", perm.size(0),
              ", where size_k = ", size_k);

  // Detect groupsize and act_order
  int num_groups = -1;
  int group_size = -1;
  bool has_act_order = g_idx.size(0) != 0;

  int b_rank = b_scales.sizes().size();
  TORCH_CHECK(b_rank == 2, "b_scales rank = ", b_rank, " is not 2");
  TORCH_CHECK(b_scales.size(1) == size_n, "b_scales dim 1 = ", b_scales.size(1),
              " is not size_n = ", size_n);
  num_groups = b_scales.size(0);

  if (has_act_order) {
    if (is_k_full) {
      TORCH_CHECK(num_groups > 1, "For act_order, num_groups must be > 1");
      TORCH_CHECK(size_k % num_groups == 0, "size_k = ", size_k,
                  ", is not divisible by num_groups = ", num_groups);
      group_size = size_k / num_groups;
    } else {
      group_size = 0;
    }

  } else {
    if (num_groups > 1) {
      TORCH_CHECK(
          size_k % num_groups == 0, "size_k = ", size_k,
          ", is not divisible by b_scales.size(0) = ", b_scales.size(0));
      group_size = size_k / num_groups;
    } else {
      group_size = -1;
    }
  }

  // Verify workspace size
  TORCH_CHECK(
      size_n % gptq_marlin::min_thread_n == 0, "size_n = ", size_n,
      ", is not divisible by min_thread_n = ", gptq_marlin::min_thread_n);
  int min_workspace_size =
      (size_n / gptq_marlin::min_thread_n) * gptq_marlin::max_par;
  TORCH_CHECK(workspace.numel() >= min_workspace_size,
              "workspace.numel = ", workspace.numel(),
              " is below min_workspace_size = ", min_workspace_size);

  int dev = a.get_device();
  if (a.scalar_type() == at::ScalarType::Half) {
    gptq_marlin::fp16::marlin_mm_f16i4(
        a.data_ptr<at::Half>(), b_q_weight.data_ptr(), c.data_ptr<at::Half>(), b_scales.data_ptr<at::Half>(),
        g_idx.data_ptr(), perm.data_ptr(), a_tmp.data_ptr<at::Half>(), size_m, size_n,
        size_k, workspace.data_ptr(), num_bits, has_act_order, is_k_full,
        num_groups, group_size, dev, at::cuda::getCurrentCUDAStream(dev),
        thread_k, thread_n, sms, gptq_marlin::max_par);
  } else if (a.scalar_type() == at::ScalarType::BFloat16) {
    gptq_marlin::bf16::marlin_mm_f16i4(
        a.data_ptr<at::BFloat16>(), b_q_weight.data_ptr(), c.data_ptr<at::BFloat16>(), b_scales.data_ptr<at::BFloat16>(),
        g_idx.data_ptr(), perm.data_ptr(), a_tmp.data_ptr<at::BFloat16>(), size_m, size_n,
        size_k, workspace.data_ptr(), num_bits, has_act_order, is_k_full,
        num_groups, group_size, dev, at::cuda::getCurrentCUDAStream(dev),
        thread_k, thread_n, sms, gptq_marlin::max_par);
  } else {
    throw std::runtime_error("gpt_marlin_gemm only supports bfloat16 and float16");
  }

  return c;
}

#endif
