#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

template<typename scalar_t>
__global__ void scaled_fp8_quant_kernel(
  c10::Float8_e4m3fn* __restrict__ out,
  const scalar_t* __restrict__ input,
  const float* __restrict__ scale,
  int64_t num_elems) {
  __shared__ float cache[1024];
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int cacheIndex = threadIdx.x;

  scalar_t tmp = 0.0;
  while (i < num_elems) {
    float x = static_cast<float>(input[i]);
    tmp = max(tmp, fabs(x));
    i += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = tmp;

  __syncthreads();

  // perform parallel reduction
  int ib = blockDim.x / 2;
  while (ib != 0) {
    if (cacheIndex < ib && cache[cacheIndex + ib] > cache[cacheIndex]) {
        cache[cacheIndex] = cache[cacheIndex + ib];
    }
    __syncthreads();
    ib /= 2;
  }
  // now cache[0] contains the maximum, rescale the numbers
  i = blockDim.x * blockIdx.x + threadIdx.x;
  while (i < num_elems) {
    out[i] = static_cast<c10::Float8_e4m3fn>(input[i] / cache[0]);
    i += blockDim.x * gridDim.x;
  }
}

} // namespace vllm

void scaled_fp8_quant(
  torch::Tensor& out,      // [..., d]
  torch::Tensor& input,    // [..., d]
  torch::Tensor& scales)   // [d]
{
  int64_t num_elems = input.numel();
  dim3 grid(1);
  dim3 block(1024);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    input.scalar_type(),
    "scaled_fp8_quant_kernel",
    [&] {
      vllm::scaled_fp8_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<c10::Float8_e4m3fn>(),
        input.data_ptr<scalar_t>(),
        scales.data_ptr<float>(),
        num_elems);
      });
}