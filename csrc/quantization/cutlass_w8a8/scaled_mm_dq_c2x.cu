#include <stddef.h>
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>

// clang-format will break include orders
// clang-format off
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/util/device_memory.h"

#include "cutlass/cutlass.h"
#include "cutlass/gemm_coord.h"
#include "cutlass/arch/mma_sm75.h"
#include "cutlass/arch/arch.h"
#include "cutlass/arch/mma.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"

#include "cutlass/epilogue/threadblock/fusion/visitors.hpp"
#include "cutlass/gemm/kernel/default_gemm_universal_with_visitor.h"

#include "broadcast_load_epilogue_c2x.hpp"
#include "common.hpp"
// clang-format on

using namespace cute;

/*
   This defines a quantized GEMM operation with dequantized output, similar to
   torch._scaled_mm. It is defined using the CUTLASS 2.x API, and is used for
   NVIDIA GPUs with SM versions prior to sm90 (Hopper).

   A and B may be both either int8 or fp8_e4m3. A can be quantized per-tensor or
   per-row. B can be quantized per-tensor or per-column.
   Any combination of per-tensor and per-row or column is supported.
   A and B must have symmetric quantization (zero point == 0).

   So the GEMM operation is D = (a_scales * A) (b_scales * B), where the
   scales are applied elementwise with numpy-style broadcasting.

   ScaleA and ScaleB define the epilogue functions that apply the scales for
   the A and B operands respectively. These scales may be either per-tensor or
   per row or column.
*/

namespace {

template <typename ElementD, typename OutputTileThreadMap>
struct TrivialEpilogue {
 private:
  using Accum = cutlass::epilogue::threadblock::VisitorAccFetch;

 public:
  using EVTCompute = cutlass::epilogue::threadblock::Sm80EVT<Accum>;
  using ArgumentType = typename EVTCompute::Arguments;

  template <typename... Args>
  static ArgumentType prepare_args(Args... args) {
    return {};
  }
};

template <typename ElementD, typename OutputTileThreadMap>
struct ScaledEpilogue {
 private:
  using Accum = cutlass::epilogue::threadblock::VisitorAccFetch;

  using ScaleA = cutlass::epilogue::threadblock::VisitorColOrScalarBroadcast<
      OutputTileThreadMap, float, Stride<Int<1>, Int<0>, Int<0>>>;

  using ScaleB = cutlass::epilogue::threadblock::VisitorRowOrScalarBroadcast<
      OutputTileThreadMap, float, Stride<Int<0>, Int<1>, Int<0>>>;

  using Compute0 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, float, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 =
      cutlass::epilogue::threadblock::Sm80EVT<Compute0, ScaleB, Accum>;

  using Compute1 = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, ElementD, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

 public:
  using EVTCompute =
      cutlass::epilogue::threadblock::Sm80EVT<Compute1, ScaleA, EVTCompute0>;
  using ArgumentType = typename EVTCompute::Arguments;

  template <typename... Args>
  static ArgumentType prepare_args(Args... args) {
    auto tuple = std::make_tuple(args...);

    torch::Tensor const& a_scales = std::get<0>(tuple);
    torch::Tensor const& b_scales = std::get<1>(tuple);

    auto a_scales_ptr = a_scales.data_ptr<float>();
    auto b_scales_ptr = b_scales.data_ptr<float>();

    using ScaleAArgs = typename ScaleA::Arguments;
    using ScaleBArgs = typename ScaleB::Arguments;

    ScaleBArgs b_args{b_scales.data_ptr<float>(), b_scales.numel() != 1, {}};
    ScaleAArgs a_args{a_scales.data_ptr<float>(), a_scales.numel() != 1, {}};

    typename EVTCompute0::Arguments evt0_compute_args{b_args};

    typename EVTCompute::Arguments evt_compute_args{a_args, evt0_compute_args};
    return evt_compute_args;
  }
};

template <typename Arch, typename ElementAB_, typename ElementD_,
          template <typename, typename> typename Epilogue_, typename TileShape,
          typename WarpShape, typename InstructionShape, int32_t MainLoopStages>
struct cutlass_2x_gemm {
  using ElementAB = ElementAB_;
  using ElementD = ElementD_;

  using ElementAcc =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>, int32_t,
                                float>::type;

  using Operator =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>,
                                cutlass::arch::OpMultiplyAddSaturate,
                                cutlass::arch::OpMultiplyAdd>::type;

  using OutputTileThreadMap =
      cutlass::epilogue::threadblock::OutputTileThreadLayout<
          TileShape, WarpShape, float, 4, 1 /* epilogue stages */
          >;

  using Epilogue = Epilogue_<ElementD, OutputTileThreadMap>;
  using EVTCompute = typename Epilogue::EVTCompute;

  using D = cutlass::epilogue::threadblock::VisitorAuxStore<
      OutputTileThreadMap, ElementD, cutlass::FloatRoundStyle::round_to_nearest,
      Stride<int64_t, Int<1>, Int<0>>>;

  using EVTD = cutlass::epilogue::threadblock::Sm80EVT<D, EVTCompute>;

  // clang-format off
  using RowMajor = typename cutlass::layout::RowMajor;
  using ColumnMajor = typename cutlass::layout::ColumnMajor;
  using KernelType = 
    typename cutlass::gemm::kernel::DefaultGemmWithVisitor<
      ElementAB, RowMajor, cutlass::ComplexTransform::kNone, 16, 
      ElementAB, ColumnMajor, cutlass::ComplexTransform::kNone, 16, 
      float, cutlass::layout::RowMajor, 4,
      ElementAcc, float, cutlass::arch::OpClassTensorOp, 
      Arch, 
      TileShape, WarpShape, InstructionShape,
      EVTD,
      cutlass::gemm::threadblock::ThreadblockSwizzleStreamK,
      MainLoopStages, Operator,
      1 /* epilogue stages */
      >::GemmKernel;
  // clang-format on

  using Op = cutlass::gemm::device::GemmUniversalAdapter<KernelType>;
};

template <typename Gemm, typename... EpilogueArgs>
void cutlass_gemm_caller(torch::Tensor& out, torch::Tensor const& a,
                         torch::Tensor const& b,
                         EpilogueArgs&&... epilogue_params) {
  using ElementAB = typename Gemm::ElementAB;
  using ElementD = typename Gemm::ElementD;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);
  cutlass::gemm::GemmCoord problem_size{m, n, k};

  int64_t lda = a.stride(0);
  int64_t ldb = b.stride(1);
  int64_t ldc = out.stride(0);

  using StrideC = Stride<int64_t, Int<1>, Int<0>>;
  StrideC c_stride{ldc, Int<1>{}, Int<0>{}};

  auto a_ptr = static_cast<ElementAB const*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB const*>(b.data_ptr());
  auto c_ptr = static_cast<ElementD*>(out.data_ptr());

  typename Gemm::D::Arguments d_args{c_ptr, c_stride};

  using Epilogue = typename Gemm::Epilogue;
  auto evt_args =
      Epilogue::prepare_args(std::forward<EpilogueArgs>(epilogue_params)...);

  typename Gemm::EVTD::Arguments epilogue_args{
      evt_args,
      d_args,
  };

  typename Gemm::Op::Arguments args{
      cutlass::gemm::GemmUniversalMode::kGemmSplitKParallel,  // universal mode
      problem_size,                                           // problem size
      1,                                                      // batch count
      epilogue_args,
      a_ptr,
      b_ptr,
      nullptr,
      nullptr,
      0,
      0,
      0,
      0,
      lda,
      ldb,
      ldc,
      ldc};

  // Launch the CUTLASS GEMM kernel.
  typename Gemm::Op gemm_op;
  size_t workspace_size = gemm_op.get_workspace_size(args);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  CUTLASS_CHECK(gemm_op.can_implement(args));
  cutlass::Status status = gemm_op(args, workspace.get(), stream);
  CUTLASS_CHECK(status);
}

}  // namespace

void cutlass_scaled_mm_sm75(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<8, 8, 16>;

  if (out.dtype() == torch::kInt8) {
    return cutlass_gemm_caller<
        cutlass_2x_gemm<cutlass::arch::Sm75, int8_t, int8_t, ScaledEpilogue,
                        TileShape, WarpShape, InstructionShape, 2>>(
        out, a, b, a_scales, b_scales);
  } else if (out.dtype() == torch::kBFloat16) {
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm75, int8_t, cutlass::bfloat16_t, ScaledEpilogue,
        TileShape, WarpShape, InstructionShape, 2>>(out, a, b, a_scales,
                                                    b_scales);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm75, int8_t, cutlass::half_t, ScaledEpilogue, TileShape,
        WarpShape, InstructionShape, 2>>(out, a, b, a_scales, b_scales);
  }
}

void cutlass_scaled_mm_sm80(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;

  if (out.dtype() == torch::kInt8) {
    return cutlass_gemm_caller<
        cutlass_2x_gemm<cutlass::arch::Sm80, int8_t, int8_t, ScaledEpilogue,
                        TileShape, WarpShape, InstructionShape, 5>>(
        out, a, b, a_scales, b_scales);
  } else if (out.dtype() == torch::kBFloat16) {
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm80, int8_t, cutlass::bfloat16_t, ScaledEpilogue,
        TileShape, WarpShape, InstructionShape, 5>>(out, a, b, a_scales,
                                                    b_scales);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm80, int8_t, cutlass::half_t, ScaledEpilogue, TileShape,
        WarpShape, InstructionShape, 5>>(out, a, b, a_scales, b_scales);
  }
}

void cutlass_scaled_mm_sm89(torch::Tensor& out, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales) {
  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;

  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(b.dtype() == torch::kInt8);

    if (out.dtype() == torch::kInt8) {
      return cutlass_gemm_caller<
          cutlass_2x_gemm<cutlass::arch::Sm89, int8_t, int8_t, ScaledEpilogue,
                          TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, a_scales, b_scales);
    } else if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, int8_t, cutlass::bfloat16_t, ScaledEpilogue,
          TileShape, WarpShape, InstructionShape, 5>>(out, a, b, a_scales,
                                                      b_scales);
    } else {
      assert(out.dtype() == torch::kFloat16);
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, int8_t, cutlass::half_t, ScaledEpilogue,
          TileShape, WarpShape, InstructionShape, 5>>(out, a, b, a_scales,
                                                      b_scales);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kFloat8_e4m3fn) {
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, cutlass::float_e4m3_t, cutlass::float_e4m3_t,
          ScaledEpilogue, TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, a_scales, b_scales);
    } else if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, cutlass::float_e4m3_t, cutlass::bfloat16_t,
          ScaledEpilogue, TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, a_scales, b_scales);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, cutlass::float_e4m3_t, cutlass::half_t,
          ScaledEpilogue, TileShape, WarpShape, InstructionShape, 5>>(
          out, a, b, a_scales, b_scales);
    }
  }
}

void cutlass_gemm_sm75(torch::Tensor& out, torch::Tensor const& a,
                       torch::Tensor const& b) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<8, 8, 16>;

  if (out.dtype() == torch::kInt8) {
    return cutlass_gemm_caller<
        cutlass_2x_gemm<cutlass::arch::Sm75, int8_t, int8_t, TrivialEpilogue,
                        TileShape, WarpShape, InstructionShape, 2>>(out, a, b);
  } else if (out.dtype() == torch::kBFloat16) {
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm75, int8_t, cutlass::bfloat16_t, TrivialEpilogue,
        TileShape, WarpShape, InstructionShape, 2>>(out, a, b);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm75, int8_t, cutlass::half_t, TrivialEpilogue,
        TileShape, WarpShape, InstructionShape, 2>>(out, a, b);
  }
}

void cutlass_gemm_sm80(torch::Tensor& out, torch::Tensor const& a,
                       torch::Tensor const& b) {
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;

  if (out.dtype() == torch::kInt8) {
    return cutlass_gemm_caller<
        cutlass_2x_gemm<cutlass::arch::Sm80, int8_t, int8_t, TrivialEpilogue,
                        TileShape, WarpShape, InstructionShape, 5>>(out, a, b);
  } else if (out.dtype() == torch::kBFloat16) {
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm80, int8_t, cutlass::bfloat16_t, TrivialEpilogue,
        TileShape, WarpShape, InstructionShape, 5>>(out, a, b);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_gemm_caller<cutlass_2x_gemm<
        cutlass::arch::Sm80, int8_t, cutlass::half_t, TrivialEpilogue,
        TileShape, WarpShape, InstructionShape, 5>>(out, a, b);
  }
}

void cutlass_gemm_sm89(torch::Tensor& out, torch::Tensor const& a,
                       torch::Tensor const& b) {
  using TileShape = typename cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape = typename cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = typename cutlass::gemm::GemmShape<16, 8, 32>;

  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(b.dtype() == torch::kInt8);

    if (out.dtype() == torch::kInt8) {
      return cutlass_gemm_caller<
          cutlass_2x_gemm<cutlass::arch::Sm89, int8_t, int8_t, TrivialEpilogue,
                          TileShape, WarpShape, InstructionShape, 5>>(out, a,
                                                                      b);
    } else if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, int8_t, cutlass::bfloat16_t, TrivialEpilogue,
          TileShape, WarpShape, InstructionShape, 5>>(out, a, b);
    } else {
      assert(out.dtype() == torch::kFloat16);
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, int8_t, cutlass::half_t, TrivialEpilogue,
          TileShape, WarpShape, InstructionShape, 5>>(out, a, b);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kFloat8_e4m3fn) {
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, cutlass::float_e4m3_t, cutlass::float_e4m3_t,
          TrivialEpilogue, TileShape, WarpShape, InstructionShape, 5>>(out, a,
                                                                       b);
    } else if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, cutlass::float_e4m3_t, cutlass::bfloat16_t,
          TrivialEpilogue, TileShape, WarpShape, InstructionShape, 5>>(out, a,
                                                                       b);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_caller<cutlass_2x_gemm<
          cutlass::arch::Sm89, cutlass::float_e4m3_t, cutlass::half_t,
          TrivialEpilogue, TileShape, WarpShape, InstructionShape, 5>>(out, a,
                                                                       b);
    }
  }
}
