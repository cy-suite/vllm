#include "hip/hip_runtime.h"
// clang-format will break include orders
// clang-format off
#include <cudaTypedefs.h>

#if defined CUDA_VERSION && CUDA_VERSION >= 12000

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>

#include <iostream>
#include <sstream>
#include <vector>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/util/device_memory.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"

#include "broadcast_load_epilogue_c3x.hpp"
#include "common.hpp"
// clang-format on

using namespace cute;

/*
   This defines a quantized GEMM operation with dequantized output, similar to
   torch._scaled_mm. It is defined using the CUTLASS 3.x API, and is used for
   NVIDIA GPUs with sm90a (Hopper) or later.

   A and B may be both either int8 or fp8_e4m3. A can be quantized per-tensor or
   per-row. B can be quantized per-tensor or per-column.
   Any combination of per-tensor and per-row or column is supported.
   A and B must have symmetric quantization (zero point == 0).

   D = a_scales * (b_scales * (A*B)) + per-row bias

   In the epilogue ACC stores the results of A*B and will be multiplied
   by a_scales and b_scales before adding the per-row bias.

   The epilogue computation can be composed with `multiplies` and `multiply_add`

   ScaleA and ScaleB define the epilogue functions that apply the scales for
   the A and B operands respectively. These scales may be either per-tensor or
   per row or column.
*/

namespace {

uint32_t next_pow_2(uint32_t const num) {
  if (num <= 1) return num;
  return 1 << (CHAR_BIT * sizeof(num) - __builtin_clz(num - 1));
}

// A wrapper for the GEMM kernel that is used to guard against compilation on
// architectures that will never use the kernel. The purpose of this is to
// reduce the size of the compiled binary.
// __CUDA_ARCH__ is not defined in host code, so this lets us smuggle the ifdef
// into code that will be executed on the device where it is defined.
template <typename Kernel>
struct enable_sm90_or_later : Kernel {
  template <typename... Args>
  CUTLASS_DEVICE void operator()(Args&&... args) {
  #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 900
    Kernel::operator()(std::forward<Args>(args)...);
  #endif
  }
};

template <typename ElementAB_, typename ElementD_, typename TileShape,
          typename ClusterShape, typename KernelSchedule,
          typename EpilogueSchedule>
struct cutlass_3x_gemm_bias {
  using ElementAB = ElementAB_;
  using ElementD = ElementD_;
  using ElementAcc =
      typename std::conditional<std::is_same_v<ElementAB, int8_t>, int32_t,
                                float>::type;
  //   using ElementBias = float;
  using ElementBias =
      typename std::conditional<std::is_same_v<ElementD, cutlass::half_t>,
                                at::Half, at::BFloat16>::type;

  using EpilogueDescriptor =
      cutlass::epilogue::collective::detail::EpilogueDescriptor<
          TileShape, cutlass::epilogue::collective::EpilogueTileAuto, ElementD,
          ElementD, EpilogueSchedule>;

  // D = a_scales * (b_scales * (A*B)) + per-row bias
  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using ScaleA = cutlass::epilogue::fusion::Sm90ColOrScalarBroadcast<
      0 /*Stages*/, typename EpilogueDescriptor::TileShape, float,
      Stride<Int<1>, Int<0>, Int<0>>>;

  using ScaleBDescriptor =
      cutlass::epilogue::collective::detail::RowBroadcastDescriptor<
          EpilogueDescriptor, float>;

  using ScaleB = cutlass::epilogue::fusion::Sm90RowOrScalarBroadcast<
      ScaleBDescriptor::Stages, typename EpilogueDescriptor::TileShape,
      typename ScaleBDescriptor::Element, Stride<Int<0>, Int<1>, Int<0>>>;

  // binary op
  using Compute0 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies, float, float,
      cutlass::FloatRoundStyle::round_to_nearest>;

  // b_scales * (A*B)
  using EVTCompute0 =
      cutlass::epilogue::fusion::Sm90EVT<Compute0, ScaleB, Accum>;

  using Compute1 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiply_add, ElementD /*ElementOutput*/,
      float /*ElementCompute*/, cutlass::FloatRoundStyle::round_to_nearest>;

  using BiasDescriptor =
      cutlass::epilogue::collective::detail::RowBroadcastDescriptor<
          EpilogueDescriptor, ElementBias>;

  using Bias = cutlass::epilogue::fusion::Sm90RowOrScalarBroadcast<
      BiasDescriptor::Stages, typename EpilogueDescriptor::TileShape,
      typename BiasDescriptor::Element, Stride<Int<0>, Int<1>, Int<0>>>;

  // a_scales * (b_scales * (A*B)) + per-row bias
  using EVTCompute1 =
      cutlass::epilogue::fusion::Sm90EVT<Compute1, ScaleA, EVTCompute0, Bias>;

  using StrideD = Stride<int64_t, Int<1>, Int<0>>;
  using ElementC = void;
  using StrideC = StrideD;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp, TileShape,
          ClusterShape, cutlass::epilogue::collective::EpilogueTileAuto,
          ElementAcc, float, ElementC, StrideC, 4, ElementD, StrideD, 4,
          EpilogueSchedule, EVTCompute1>::CollectiveOp;

  static constexpr size_t CEStorageSize =
      sizeof(typename CollectiveEpilogue::SharedStorage);
  using Stages = typename cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(CEStorageSize)>;

  // clang-format off
  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp, 
          ElementAB, cutlass::layout::RowMajor, 16, 
          ElementAB, cutlass::layout::ColumnMajor, 16, 
          ElementAcc, TileShape, ClusterShape,
          Stages,
          KernelSchedule>::CollectiveOp;
  // clang-format on

  using KernelType = enable_sm90_or_later<cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue,
      cutlass::gemm::PersistentScheduler>>;

  struct GemmKernel : public KernelType {};
};

template <typename Gemm>
void cutlass_scaled_mm_bias_dispatcher(torch::Tensor& out,
                                          torch::Tensor const& a,
                                          torch::Tensor const& b,
                                          torch::Tensor const& a_scales,
                                          torch::Tensor const& b_scales,
                                          torch::Tensor const& bias) {
  using ElementAB = typename Gemm::ElementAB;
  using ElementD = typename Gemm::ElementD;
  using ElementBias = typename Gemm::ElementBias;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);

  int64_t lda = a.stride(0);
  int64_t ldb = b.stride(1);
  int64_t ldc = out.stride(0);

  using StrideA = Stride<int64_t, Int<1>, Int<0>>;
  using StrideB = Stride<int64_t, Int<1>, Int<0>>;
  using StrideC = typename Gemm::StrideC;

  StrideA a_stride{lda, Int<1>{}, Int<0>{}};
  StrideB b_stride{ldb, Int<1>{}, Int<0>{}};
  StrideC c_stride{ldc, Int<1>{}, Int<0>{}};

  using GemmKernel = typename Gemm::GemmKernel;
  typename GemmKernel::ProblemShape prob_shape{m, n, k, 1};

  auto a_ptr = static_cast<ElementAB*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB*>(b.data_ptr());
  typename GemmKernel::MainloopArguments mainloop_args{a_ptr, a_stride, b_ptr,
                                                       b_stride};

  auto c_ptr = static_cast<ElementD*>(out.data_ptr());
  typename GemmKernel::EpilogueArguments epilogue_args{
      {} /* epilogue.thread */, c_ptr, c_stride, c_ptr, c_stride};

  typename GemmKernel::Arguments args{cutlass::gemm::GemmUniversalMode::kGemm,
                                      prob_shape, mainloop_args, epilogue_args};

  using ScaleA_Args = typename Gemm::ScaleA::Arguments;
  using ScaleB_Args = typename Gemm::ScaleB::Arguments;
  using Bias_Args = typename Gemm::Bias::Arguments;
  ScaleA_Args a_args{a_scales.data_ptr<float>(), a_scales.numel() != 1, {}};
  ScaleB_Args b_args{b_scales.data_ptr<float>(), b_scales.numel() != 1, {}};
  Bias_Args bias_args =
      Bias_Args{bias.data_ptr<ElementBias>(), bias.numel() != 1, {}};

  args.epilogue.thread = {
      // ternary op: a_scales * (b_scales * (A*B)) + per-row bias
      a_args,  // a_scales
      {
          // binary op: b_scales * (A*B)
          b_args,  // b_scales
          {},      // acc
          {}       // binary args: multiplies
      },
      bias_args,  // bias
      {}          // ternary args: multiply_add
  };

  // Launch the CUTLASS GEMM kernel.
  using GemmOp = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  GemmOp gemm_op;
  CUTLASS_CHECK(gemm_op.can_implement(args));

  size_t workspace_size = gemm_op.get_workspace_size(args);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  cutlass::Status status = gemm_op.run(args, workspace.get(), stream);
  CUTLASS_CHECK(status);
}

template <typename InType, typename OutType, int32_t M>
struct sm90_fp8_config {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_2, _1, _1>;

  using Cutlass3xGemm =
      cutlass_3x_gemm_bias<InType, OutType, TileShape, ClusterShape,
                           KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType>
struct sm90_fp8_config<InType, OutType, 128> {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _128, _128>;
  using ClusterShape = Shape<_2, _1, _1>;

  using Cutlass3xGemm =
      cutlass_3x_gemm_bias<InType, OutType, TileShape, ClusterShape,
                           KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType>
struct sm90_fp8_config<InType, OutType, 64> {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_64, _64, _128>;
  using ClusterShape = Shape<_1, _8, _1>;

  using Cutlass3xGemm =
      cutlass_3x_gemm_bias<InType, OutType, TileShape, ClusterShape,
                           KernelSchedule, EpilogueSchedule>;
};

}  // namespace

template <typename InType, typename OutType>
void cutlass_scaled_mm_bias_sm90_fp8_dispatch(torch::Tensor& out,
                                                 torch::Tensor const& a,
                                                 torch::Tensor const& b,
                                                 torch::Tensor const& a_scales,
                                                 torch::Tensor const& b_scales,
                                                 torch::Tensor const& bias) {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  TORCH_CHECK((std::is_same_v<OutType, cutlass::half_t> &&
               (bias.dtype() == torch::kFloat16)) ||
              (std::is_same_v<OutType, cutlass::bfloat16_t> &&
               (bias.dtype() == torch::kBFloat16)));

  using Cutlass3xGemmDefault =
      typename sm90_fp8_config<InType, OutType, 0>::Cutlass3xGemm;
  using Cutlass3xGemmM64 =
      typename sm90_fp8_config<InType, OutType, 64>::Cutlass3xGemm;
  using Cutlass3xGemmM128 =
      typename sm90_fp8_config<InType, OutType, 128>::Cutlass3xGemm;

  uint32_t const m = a.size(0);
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(64), next_pow_2(m));  // next power of 2

  if (mp2 <= 64) {
    // m in [1, 64]
    return cutlass_scaled_mm_bias_dispatcher<Cutlass3xGemmM64>(
        out, a, b, a_scales, b_scales, bias);
  } else if (mp2 <= 128) {
    // m in (64, 128]
    return cutlass_scaled_mm_bias_dispatcher<Cutlass3xGemmM128>(
        out, a, b, a_scales, b_scales, bias);
  } else {
    // m in (128, inf)
    return cutlass_scaled_mm_bias_dispatcher<Cutlass3xGemmDefault>(
        out, a, b, a_scales, b_scales, bias);
  }
}

void cutlass_scaled_mm_bias_sm90(torch::Tensor& out, torch::Tensor const& a,
                                    torch::Tensor const& b,
                                    torch::Tensor const& a_scales,
                                    torch::Tensor const& b_scales,
                                    torch::Tensor const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(out.dtype() == bias.dtype());
  TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

  if (out.dtype() == torch::kBFloat16) {
    return cutlass_scaled_mm_bias_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                                       cutlass::bfloat16_t>(
        out, a, b, a_scales, b_scales, bias);
  } else {
    TORCH_CHECK(out.dtype() == torch::kFloat16);
    return cutlass_scaled_mm_bias_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                                       cutlass::half_t>(
        out, a, b, a_scales, b_scales, bias);
  }
}

#endif
