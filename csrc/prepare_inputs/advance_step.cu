#include "hip/hip_runtime.h"
/*
 * TODO: Add doc
 */

#include "advance_step.cuh"

namespace prepare_inputs {

template <int const num_threads>
__global__ void advance_step_kernel(int num_seqs, int block_size,
                                    long const* sampled_token_ids_ptr,
                                    long* input_positions_ptr,
                                    int* seq_lens_ptr, int* slot_mapping_ptr,
                                    int const* block_tables_ptr,
                                    int64_t const block_tables_stride) {
  int num_seq_blocks = div_ceil(num_seqs, num_threads);

  if (blockIdx.x > num_seq_blocks) {
    return;
  }

  int cur_seq_id = blockIdx.x * num_threads + threadIdx.x;

  if (cur_seq_id > num_seqs) {
    return;
  }

  int seq_len = seq_lens_ptr[cur_seq_id];
  int next_seq_len = seq_len + 1;
  int next_input_pos = seq_len;

  seq_lens_ptr[cur_seq_id] = next_seq_len;
  input_positions_ptr[cur_seq_id] = next_input_pos;

  int const* seq_block_tables_ptr =
      block_tables_ptr + block_tables_stride * cur_seq_id;

  int block_index = next_input_pos / block_size;
  int block_offset = next_input_pos % block_size;

  int slot_num = seq_block_tables_ptr[block_index] * block_size + block_offset;
  slot_mapping_ptr[cur_seq_id] = slot_num;
}

inline void verify_tensor(torch::Tensor& t, int64_t const size_0,
                          int64_t const size_1, c10::ScalarType const type) {
  if (size_0 != -1) {
    TORCH_CHECK(t.size(0) == size_0, "Shape mismatch: t.size(0) = ", t.size(0),
                ", size_0 = ", size_0);
  }

  if (size_1 != -1) {
    TORCH_CHECK(t.size(1) == size_1, "Shape mismatch: t.size(1) = ", t.size(1),
                ", size_1 = ", size_1);
  }

  TORCH_CHECK(t.is_contiguous(), "Not contiguous");
  TORCH_CHECK(t.dtype() == type, "Type is not ", type);
}

void advance_step(int num_seqs, int block_size,
                  torch::Tensor& sampled_token_ids,  // type: long
                  torch::Tensor& input_positions,    // type: long
                  torch::Tensor& seq_lens,           // type: int
                  torch::Tensor& slot_mapping,       // type: long
                  torch::Tensor& block_tables) {     // type: int
  // Verify all tensors
  verify_tensor(sampled_token_ids, num_seqs, 1, at::kLong);
  verify_tensor(input_positions, num_seqs, -1, at::kLong);
  verify_tensor(seq_lens, num_seqs, -1, at::kInt);
  verify_tensor(slot_mapping, num_seqs, -1, at::kLong);
  verify_tensor(block_tables, num_seqs, -1, at::kLong);

  int dev = sampled_token_ids.get_device();
  hipStream_t stream = at::cuda::getCurrentCUDAStream(dev);

  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  advance_step_kernel<max_threads><<<blocks, max_threads, 0, stream>>>(
      num_seqs, block_size,
      reinterpret_cast<long const*>(sampled_token_ids.data_ptr()),
      reinterpret_cast<long*>(input_positions.data_ptr()),
      reinterpret_cast<int*>(seq_lens.data_ptr()),
      reinterpret_cast<int*>(slot_mapping.data_ptr()),
      reinterpret_cast<int const*>(block_tables.data_ptr()),
      block_tables.stride(0));
}

}  // namespace prepare_inputs

void advance_step(int num_seqs, int block_size,
                  torch::Tensor& sampled_token_ids,
                  torch::Tensor& input_positions, torch::Tensor& seq_lens,
                  torch::Tensor& slot_mapping, torch::Tensor& block_tables) {
  prepare_inputs::advance_step(num_seqs, block_size, sampled_token_ids,
                               input_positions, seq_lens, slot_mapping,
                               block_tables);
}