#include "hip/hip_runtime.h"
/*
 * TODO: Add doc
 */

#include "advance_step.cuh"

namespace prepare_inputs {

template <int const num_threads>
__global__ void advance_step_kernel(int num_seqs, int block_size,
                                    long const* sampled_token_ids_ptr,
                                    long* input_positions_ptr,
                                    int* seq_lens_ptr, int* slot_mapping_ptr,
                                    int const* block_tables_ptr,
                                    int64_t const block_tables_stride) {
  int num_seq_blocks = div_ceil(num_seqs, num_threads);

  if (blockIdx.x >= num_seq_blocks) {
    return;
  }

  int cur_seq_id = blockIdx.x * num_threads + threadIdx.x;

  if (cur_seq_id >= num_seqs) {
    return;
  }

  int seq_len = seq_lens_ptr[cur_seq_id];
  int next_seq_len = seq_len + 1;
  int next_input_pos = next_seq_len - 1;

  seq_lens_ptr[cur_seq_id] = next_seq_len;
  input_positions_ptr[cur_seq_id] = next_input_pos;

  int const* seq_block_tables_ptr =
      block_tables_ptr + block_tables_stride * cur_seq_id;

  int block_index = next_input_pos / block_size;
  int block_offset = next_input_pos % block_size;

  int slot_num = seq_block_tables_ptr[block_index] * block_size + block_offset;
  slot_mapping_ptr[cur_seq_id] = slot_num;
}

inline void verify_tensor(std::string const& name, torch::Tensor& t,
                          int64_t const size_0, int64_t const size_1,
                          c10::ScalarType const type) {
  bool size_0_cond = true;
  if (size_0 != -1) {
    size_0_cond = t.size(0) == size_0;
  }

  bool size_1_cond = true;
  if (size_1 != -1) {
    size_1_cond = t.size(1) == size_1;
  }

  bool is_contiguous = t.is_contiguous();
  bool same_type = t.dtype() == type;

  bool pass = size_0_cond && size_1_cond && is_contiguous && same_type;
  if (!pass) {
    TORCH_CHECK(false, "tensor: name = ", name, ", shape = ", t.sizes(),
                " is_cont = ", t.is_contiguous(), ", type = ", t.dtype(),
                " is not as expected: shape = [", size_0, ", ", size_1,
                "], type = ", type);
  }
}

void advance_step(int num_seqs, int block_size,
                  torch::Tensor& sampled_token_ids,  // type: long
                  torch::Tensor& input_positions,    // type: long
                  torch::Tensor& seq_lens,           // type: int
                  torch::Tensor& slot_mapping,       // type: long
                  torch::Tensor& block_tables) {     // type: int

  if (logging) {
    printf("advance_step:\n");
    printf("  num_seqs = %d\n", num_seqs);
    printf("  block_size = %d\n", block_size);
  }
  // Verify all tensors
  verify_tensor("sampled_token_ids", sampled_token_ids, num_seqs, -1, at::kLong);
  verify_tensor("input_positions", input_positions, num_seqs, -1, at::kLong);
  verify_tensor("seq_lens", seq_lens, num_seqs, -1, at::kInt);
  verify_tensor("slot_mapping", slot_mapping, num_seqs, -1, at::kLong);
  verify_tensor("block_tables", block_tables, num_seqs, -1, at::kInt);

  int dev = sampled_token_ids.get_device();
  hipStream_t stream = at::cuda::getCurrentCUDAStream(dev);

  int blocks;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  advance_step_kernel<max_threads><<<blocks, max_threads, 0, stream>>>(
      num_seqs, block_size,
      reinterpret_cast<long const*>(sampled_token_ids.data_ptr()),
      reinterpret_cast<long*>(input_positions.data_ptr()),
      reinterpret_cast<int*>(seq_lens.data_ptr()),
      reinterpret_cast<int*>(slot_mapping.data_ptr()),
      reinterpret_cast<int const*>(block_tables.data_ptr()),
      block_tables.stride(0));
}

}  // namespace prepare_inputs

void advance_step(int64_t num_seqs, int64_t block_size,
                  torch::Tensor& sampled_token_ids,
                  torch::Tensor& input_positions, torch::Tensor& seq_lens,
                  torch::Tensor& slot_mapping, torch::Tensor& block_tables) {
  prepare_inputs::advance_step(num_seqs, block_size, sampled_token_ids,
                               input_positions, seq_lens, slot_mapping,
                               block_tables);
}