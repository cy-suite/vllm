#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
void cutlass_scaled_sparse_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& e,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);
#endif

bool cutlass_scaled_sparse_mm_supports_fp8(int64_t cuda_device_capability) {
  // CUTLASS FP8 kernels need at least
  //   CUDA 12.0 on SM90 systems (Hopper)
  //   CUDA 12.4 on SM89 systems (Lovelace)

#if defined CUDA_VERSION
  if (cuda_device_capability >= 90) {
    return CUDA_VERSION >= 12000;
  } else if (cuda_device_capability >= 89) {
    return CUDA_VERSION >= 12040;
  }
#endif

  return false;
}

int32_t test_get_sm_version_num() {
  int32_t major_capability, minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;
  return version_num;
}

void cutlass_scaled_sparse_mm(torch::Tensor& c, torch::Tensor const& a,
                       torch::Tensor const& e,
                       torch::Tensor const& b, torch::Tensor const& a_scales,
                       torch::Tensor const& b_scales,
                       c10::optional<torch::Tensor> const& bias) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) * 2 == b.size(0) &&
              b.size(1) == c.size(1));
  TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1 && c.stride(0) == 1); // Column-major
  // TORCH_CHECK(c.stride(0) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(b.stride(1) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous() &&
                bias->dim() == 1);
  }

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));
  int32_t version_num = test_get_sm_version_num();
  // Hopper

  // Guard against compilation issues for sm90 kernels
#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
  if (version_num >= 90) {
    cutlass_scaled_sparse_mm_sm90(c, a, e, b, a_scales, b_scales, bias);
    return;
  }
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_sparse_mm for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}
