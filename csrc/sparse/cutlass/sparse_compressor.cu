#include <cudaTypedefs.h>

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>

#include <iostream>
#include <sstream>
#include <vector>

#include "cutlass/cutlass.h"


#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/detail/dependent_false.hpp"

#include "util/broadcast_load_epilogue_c3x.hpp"
#include "util/common.hpp"

#include "cutlass/transform/device/transform_universal_adapter.hpp"
#include "cutlass/transform/kernel/sparse_gemm_compressor.hpp"

#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include <iostream>

#include "cutlass/cutlass.h"

#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/dispatch_policy.hpp"

#include "util/host_tensor.h"
#include "util/packed_stride.hpp"

#include "util/helper.h"

#include "sparse_scaled_mm_c3x.cuh"

/// Make A structured sparse by replacing elements with 0 and compress it
template<typename ElementA_>
bool sparsify_and_compress(torch::Tensor& a_compressed, torch::Tensor& e, torch::Tensor const& a)
{
  // Checks for conformality
  TORCH_CHECK(a.dtype() == torch::kInt8 ||
              a.dtype() == torch::kFloat8_e4m3fn ||
              a.dtype() == torch::kFloat16 ||
              a.dtype() == torch::kBFloat16);
  TORCH_CHECK(a.dim() == 2)
  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1)

  int m = a.size(0);
  int k = a.size(1);

  using ProblemShape = Shape<int,int,int,int>;
  using ElementA = ElementA_;
  using LayoutTagA = cutlass::layout::RowMajor;

  // Layouts for reference (non-sparse) tensors
  using StrideA = cutlass::gemm::TagToStrideA_t<LayoutTagA>;
  using StrideE = StrideA;

  using Gemm =
    typename std::conditional<std::is_same_v<ElementA, int8_t>,
      typename sm90_int8_config_default<int8_t, cutlass::half_t,
                                        ScaledEpilogue>::Cutlass3xGemm,
      typename std::conditional<std::is_same_v<ElementA, cutlass::float_e4m3_t>,
        typename sm90_fp8_config_default<cutlass::float_e4m3_t, cutlass::half_t,
                                          ScaledEpilogue>::Cutlass3xGemm,
        typename std::conditional<std::is_same_v<ElementA, cutlass::half_t>,
          typename sm90_fp16_config_default<cutlass::half_t, cutlass::half_t,
                                            ScaledEpilogue>::Cutlass3xGemm,
          typename sm90_bf16_config_default<cutlass::bfloat16_t,
                                            cutlass::half_t,
                                            ScaledEpilogue>::Cutlass3xGemm
        >::type
      >::type
    >::type;

  using ElementAB = typename Gemm::ElementAB;
  using ElementD = typename Gemm::ElementD;

  // Just a dummy value
  int32_t n = 128;

  int64_t lda = a.stride(0);

  using StrideA = Stride<int64_t, Int<1>, int64_t>;
  using StrideB = Stride<int64_t, Int<1>, int64_t>;
  using StrideC = typename Gemm::StrideC;

  StrideA a_stride{lda, Int<1>{}, 0};

  using GemmKernel = typename Gemm::GemmKernel;
  typename GemmKernel::ProblemShape prob_shape{m, n, k, 1};

  using LayoutA = typename GemmKernel::CollectiveMainloop::LayoutA;
  using LayoutE = typename GemmKernel::CollectiveMainloop::LayoutE;

  using ElementE = typename GemmKernel::CollectiveMainloop::ElementE;
  using SparseConfig = typename GemmKernel::CollectiveMainloop::SparseConfig;

  LayoutA a_layout = SparseConfig::fill_layoutA(prob_shape);
  LayoutE e_layout = SparseConfig::fill_layoutE(prob_shape);

  // typename Gemm::GemmKernel::ProblemShape prob_shape{m, 1, k, 1};

  // Offline compressor kernel
  using CompressorUtility = cutlass::transform::kernel::StructuredSparseCompressorUtility<
                            ProblemShape,
                            ElementA,
                            LayoutTagA,
                            SparseConfig>;

  using CompressorKernel = cutlass::transform::kernel::StructuredSparseCompressor<
                            ProblemShape,
                            ElementA,
                            LayoutTagA,
                            SparseConfig,
                            cutlass::arch::Sm90>;

  using Compressor = cutlass::transform::device::TransformUniversalAdapter<CompressorKernel>;
  
  auto [M, N, K, L] = prob_shape;

  StrideA stride_A;
  StrideA stride_A_compressed;
  StrideE stride_E;

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));

  CompressorUtility compressor_utility(prob_shape, stride_A);

  int ME = compressor_utility.get_metadata_m_physical();
  int KE = compressor_utility.get_metadata_k_physical();
  int KC = compressor_utility.get_tensorA_k_physical();

  auto a_ptr = static_cast<ElementA*>(a.data_ptr());

  // cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  // cutlass::DeviceAllocation<typename Gemm::ElementA> block_A_compressed;
  // cutlass::DeviceAllocation<typename Gemm::CollectiveMainloop::ElementE> block_E;

  auto a_compressed_ptr = static_cast<ElementA*>(a_compressed.data_ptr());
  auto e_ptr = static_cast<typename Gemm::CollectiveMainloop::ElementE*>(e.data_ptr());

  // block_A_compressed.reset(M * KC * L);
  // block_E.reset(ME * KE * L);

  stride_A_compressed = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, KC, L));
  stride_E = cutlass::make_cute_packed_stride(StrideE{}, cute::make_shape(ME, KE, L));

  // // Random sparsification is performed on host
  // std::vector<ElementA> block_A_host(m * k);
  // cutlass::device_memory::copy_to_host(block_A_host.data(), a_ptr, m * k);
  // compressor_utility.structure_sparse_zero_mask_fill(block_A_host.data(), 2024);
  // cutlass::device_memory::copy_to_device(a_ptr, block_A_host.data(), m * k);

  cutlass::KernelHardwareInfo hw_info;
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);
  typename Compressor::Arguments arguments {
    prob_shape,
    { a_ptr,
      stride_A,
      a_compressed_ptr,
      e_ptr },
    {hw_info} };

  Compressor compressor_op;
  size_t workspace_size = Compressor::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  CUTLASS_CHECK(compressor_op.can_implement(arguments));
  CUTLASS_CHECK(compressor_op.initialize(arguments, workspace.get()));
  CUTLASS_CHECK(compressor_op.run());
  CUDA_CHECK(hipDeviceSynchronize());

  return true;
}

bool cutlass_compress_entry(torch::Tensor& a_compressed, torch::Tensor& e, torch::Tensor const& a)
{
  if (a.dtype() == torch::kBFloat16) {
    return sparsify_and_compress<cutlass::bfloat16_t>(a_compressed, e, a);
  } else if (a.dtype() == torch::kFloat16) {
    return sparsify_and_compress<cutlass::half_t>(a_compressed, e, a);
  } else if (a.dtype() == torch::kFloat8_e4m3fn) {
    return sparsify_and_compress<cutlass::float_e4m3_t>(a_compressed, e, a);
  }
  else if (a.dtype() == torch::kInt8) {
    return sparsify_and_compress<int8_t>(a_compressed, e, a);
  }
  return false;
}