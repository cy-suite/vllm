#ifdef USE_ROCM
  #include <hip/hip_runtime.h>
  #include <hip/hip_runtime_api.h>
#endif
int64_t get_device_attribute(int64_t attribute, int64_t device_id) {
  int device, value;
  if (device_id < 0) {
    hipGetDevice(&device);
  } else {
    device = device_id;
  }
  hipDeviceGetAttribute(&value, static_cast<hipDeviceAttribute_t>(attribute),
                         device);
  return value;
}

int64_t get_max_shared_memory_per_block_device_attribute(int64_t device_id) {
  int64_t attribute;
  // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
  // hipDeviceAttributeSharedMemPerBlockOptin = 97 if not is_hip() else 74

#ifdef USE_ROCM
  attribute = hipDeviceAttributeMaxSharedMemoryPerBlock;
#else
  attribute = hipDeviceAttributeSharedMemPerBlockOptin;
#endif

  return get_device_attribute(attribute, device_id);
}

#include <torch/extension.h>
#include <vector>

torch::Tensor weak_ref_tensor(torch::Tensor tensor) {
  // Ensure tensor is on CUDA
  if (!tensor.is_cuda()) {
    throw std::runtime_error("Tensor must be on CUDA device");
  }

  // Get the raw data pointer
  void* data_ptr = tensor.data_ptr();

  // Get tensor sizes and strides
  std::vector<int64_t> sizes = tensor.sizes().vec();
  std::vector<int64_t> strides = tensor.strides().vec();

  // Get tensor options (dtype, device)
  auto options = tensor.options();

  // Create a new tensor from the raw data pointer
  auto new_tensor = torch::from_blob(data_ptr, sizes, strides, options);

  return new_tensor;
}