#include "vmm.h"

#include <c10/core/ScalarType.h>

#include <cstdint>
#include <cstdio>
#include <string>
#include <unordered_map>
#include <vector>

/*
** CacheDevicePtr functions implementation
*/

CacheDevicePtr::CacheDevicePtr()
    : dptr(0), reservedPageNum(0), allocatedPageNum(0) {}

CacheDevicePtr::~CacheDevicePtr() {
  if (dptr != 0) {
    auto status = hipMemUnmap(dptr, reservedPageNum * pageSize);
    status = hipMemAddressFree(dptr, reservedPageNum * pageSize);
  }
}

void CacheDevicePtr::setPageSize(int64_t num) { pageSize = num * 2 * _MB; }

// get hipDeviceptr_t dptr
hipDeviceptr_t CacheDevicePtr::get_dptr() { return dptr; }

// get void * type pointer
void* CacheDevicePtr::get_void_ptr() { return reinterpret_cast<void*>(dptr); }

/*
** CacheAllocator functions implementation
*/

CacheAllocator::CacheAllocator() {
  // get current device gpu id
  int currentDevice;
  auto cudaStatus = hipGetDevice(&currentDevice);
  TORCH_CHECK(cudaStatus == hipSuccess, "hipGetDevice failed!");

  // set memory allocation property struct hipMemAllocationProp,
  // which is used to control the specific behavior of memory allocation
  prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  prop.location.id = currentDevice;

  // set memory access descriptor struct hipMemAccessDesc,
  // which is used to control the access permission of memory
  accessDescr = {};
  accessDescr.location.id = prop.location.id;
  accessDescr.location.type = prop.location.type;
  accessDescr.flags = hipMemAccessFlagsProtReadWrite;
}

int64_t CacheAllocator::getGranularity() {
  hipMemGetAllocationGranularity(&granularity, &prop,
                                hipMemAllocationGranularityMinimum);
  printf("granularity: %ld Bytes\n", granularity);
  return granularity;
}

void CacheAllocator::setPageSize(int64_t num) { pageSize = num * granularity; }

// reserve function, reserve virtual address space
int64_t CacheAllocator::reserveCachePtr(
    const c10::intrusive_ptr<CacheDevicePtr>& ptr, int64_t pageNum) {
  if (pageNum == 0) {
    return hipSuccess;
  }
  size_t size = pageNum * pageSize;
  auto status = hipMemAddressReserve(&(ptr->dptr), size, 0, 0, 0);

  if (status != hipSuccess) {
    printf("hipMemAddressReserve failed! error-code: %d\n", status);
  } else {
    ptr->reservedPageNum += pageNum;
  }

  return status;
}

// alloc function, allocate physical memory, map to the reserved virtual address
// space of dptr, and set access permission
int64_t CacheAllocator::allocCachePtr(
    const c10::intrusive_ptr<CacheDevicePtr>& ptr, int64_t pageNum,
    int64_t offset) {
  if (pageNum == 0) {
    return hipSuccess;
  }
  size_t size = pageNum * pageSize;
  auto start_dptr = ptr->dptr + offset;

  hipError_t status = hipSuccess;
  hipMemGenericAllocationHandle_t allocationHandle;
  if ((status = hipMemCreate(&allocationHandle, size, &prop, 0)) ==
      hipSuccess) {
    if ((status = hipMemMap(start_dptr, size, 0, allocationHandle, 0)) ==
        hipSuccess) {
      if ((status = hipMemSetAccess(start_dptr, size, &accessDescr, 1)) ==
          hipSuccess) {
        ptr->allocatedPageNum += pageNum;
      } else {
        printf("hipMemMap success,but hipMemSetAccess failed!, err code: %d\n",
               status);
        hipMemUnmap(start_dptr, size);
      }
    }
    // always release the handle, but the memory is still can access util
    // hipMemUnmap
    hipMemRelease(allocationHandle);
  } else {
    printf("hipMemCreate failed!, err code: %d\n", status);
  }
  return status;
}

// free function, unmap the virtual address space，release physical memory
// handles and free virtual address space
int64_t CacheAllocator::freeCachePtr(
    const c10::intrusive_ptr<CacheDevicePtr>& ptr) {
  hipError_t status = hipSuccess;
  if (ptr->dptr != 0) {
    status = hipMemUnmap(ptr->dptr, ptr->reservedPageNum * pageSize);
    if (status != hipSuccess) {
      printf("hipMemUnmap failed! error-code: %d\n", status);
    } else {
      status = hipMemAddressFree(ptr->dptr, ptr->reservedPageNum * pageSize);
      if (status != hipSuccess) {
        printf("hipMemAddressFree failed! error-code: %d\n", status);
      }
    }
  }
  return status;
}

// releaseCachePtrPages function, unmap the virtual address space，release
// physical memory handles but not free virtual address space
int64_t CacheAllocator::releaseCachePtr(
    const c10::intrusive_ptr<CacheDevicePtr>& ptr, int64_t pageNum,
    int64_t offset) {
  if (pageNum == 0 && offset == 0) {
    pageNum = ptr->reservedPageNum;
  }
  if (pageNum == 0) {
    return hipSuccess;
  }
  auto start_dptr = ptr->dptr + offset;
  hipError_t status = hipSuccess;
  if (ptr->dptr != 0) {
    status = hipMemUnmap(start_dptr, pageNum * pageSize);
    if (status != hipSuccess) {
      printf("hipMemUnmap failed! error-code: %d\n", status);
    }
  }
  return status;
}

/*
** vmm other util functions implementation
*/

torch::Tensor wrap_dptr_to_tensor(hipDeviceptr_t d_ptr, const std::string dtype,
                                  at::ArrayRef<int64_t> shape) {
  // get current device gpu id
  int currentDevice;
  auto cudaStatus = hipGetDevice(&currentDevice);
  TORCH_CHECK(cudaStatus == hipSuccess, "hipGetDevice failed!");

  auto _type = c10::kFloat;

  const std::unordered_map<std::string, c10::ScalarType> typeMap = {
      // float data type
      {"float64", c10::kDouble},
      {"float32", c10::kFloat},
      {"float16", c10::kHalf},
      {"float", c10::kFloat},
      {"double", c10::kDouble},
      {"half", c10::kHalf},
      {"bfloat16", c10::kBFloat16},
      // integer data type
      {"int64", c10::kLong},
      {"int32", c10::kInt},
      {"int16", c10::kShort},
      {"int8", c10::kChar},
      {"int", c10::kInt},
      {"uint8", c10::kByte}};

  _type = typeMap.at(dtype);

  // set the data type and device of the Tensor
  auto options =
      torch::TensorOptions().dtype(_type).device(torch::kCUDA, currentDevice);

  // create a Tensor from the hipDeviceptr_t
  torch::Tensor tensor =
      torch::from_blob(reinterpret_cast<void*>(d_ptr), shape, options);

  return tensor;
}

torch::Tensor wrap_cache_ptr_to_tensor(
    const c10::intrusive_ptr<CacheDevicePtr>& ptr, const std::string dtype,
    at::ArrayRef<int64_t> shape) {
  return wrap_dptr_to_tensor(ptr->dptr, dtype, shape);
}
